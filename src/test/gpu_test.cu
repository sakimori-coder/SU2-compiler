#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>        // For STL sort
#include <thrust/sort.h>    // For thrust sort
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <time.h>

using namespace std;

struct my_type
{
    int a;
    int b;

    // __host__ __device__ my_type inline operator=(my_type other){
    //     a = other.a;
    //     b = other.b;
    //     return *this;
    // }
};


__host__ __device__ bool cmp(pair<int, int> x, pair<int, int> y){
    return x.first < y.first;
}

int main()
{
    // Creating vector with random numbers
    const int N = 100000000;
    thrust::host_vector<my_type> A_host(N), B_host(N);
    for (int i = 0; i < N; i++) {
        my_type tmp = {rand(), rand()};
        A_host[i] = tmp;
        B_host[i] = tmp;
    }

    // Data transfer (CPU -> GPU) 
    thrust::device_vector<my_type> A(N), B(N);
    thrust::copy(A_host.begin(), A_host.end(), A.begin());
    thrust::copy(B_host.begin(), B_host.end(), B.begin());


    clock_t start = clock();
    
    // Sort (thrust)
    // thrust::sort(A.begin(), A.end(), cmp);   // Sort by thrust
    // thrust::sort(B.begin(), B.end());
    clock_t end = clock();
    double time = static_cast<double>(end - start) / CLOCKS_PER_SEC * 1000.0;
    printf("time %lf[ms]\n", time);
    // Data transfer (GPU -> CPU)
    // thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    return 0;
}